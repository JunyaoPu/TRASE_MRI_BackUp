#include "hip/hip_runtime.h"
//Main simulator library.
#include "master_def.h"

//Specific coil, sequence... for this simulation.
#include <iostream>
#include "sequence/GRE.cuh"
#include "coil/coil_ideal.cuh"
#include "scanner/scanner.cuh"
#include "primitives/CylinderXY.cuh"
#include "primitives/Box.cuh"
#include "params/simuParams.cuh"
#include "util/recorder.h"
#include "util/vector3.cuh"


#include <time.h>
#include "params/TRASE_Params.cuh"

void wait ( int seconds )
{
  clock_t endwait;
  endwait = clock () + seconds * CLOCKS_PER_SEC ;
  while (clock() < endwait) {}
}


void iteration(real _num){

	//Simulation properties.
	int num_par = 102400;

	SimuParams test_params(num_par, //Number of particles.
		num_par,					//Number of particles per stream.
		8,						//Sequence repeat time.
		0.5,						//Sequence echo time.
		0.001,						//Simulation timestep.
		0,							//Number of particles to track continual, individual magnetization.
		Vector3(0, 0, 1),			//Initial magnetization vector.
		Vector3(0, 0, 0.001),		//Main B0 field direction / strength.
		65,							//(vertical) resolution.
		65,							//(horizontal) resolution.
		5,							//(vertical) FOV.
		5,							//(horizontal) FOV.
		1.005,
		_num
		);

	TRASE_Params test_TRASE(&test_params);


	Coil_Ideal test_coil;
	GRE test_sequence(&test_params);
	Scanner test_scanner(test_sequence, test_coil, test_params,test_TRASE);
	Cylinder_XY test_primitive(Vector3(0.0,0.0,0.0), 2.0,2.0, 2.0, 4.0, 0.00, 0, 0, num_par);
	test_scanner.add_primitive(test_primitive);


	//
	test_scanner.scan();
	//


	hipDeviceSynchronize();
	hipDeviceReset();


}

int main(){

	for (real i = 25; i < 26;){

		iteration(0);

		i+=100;
		wait(15);
	}


	return 0;
}
