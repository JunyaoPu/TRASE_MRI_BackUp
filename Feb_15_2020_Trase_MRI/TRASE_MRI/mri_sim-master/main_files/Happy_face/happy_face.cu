#include "hip/hip_runtime.h"
//Main simulator macro definitions.
#include "master_def.h"

//Specific coil, sequence... for this simulation.
#include <iostream>
#include "sequence/GRE.cuh"
#include "coil/coil_ideal.cuh"
#include "scanner/scanner.cuh"
#include "primitives/CylinderXY.cuh"
#include "params/simuParams.cuh"
#include "util/recorder.h"
#include "util/vector3.cuh"

#include "params/TRASE_Params.cuh"
#include <time.h>


//#include "primitives/lattice.cuh"

void wait ( int seconds )
{
  clock_t endwait;
  endwait = clock () + seconds * CLOCKS_PER_SEC ;
  while (clock() < endwait) {}
}

void iteration(double _num){

	//Simulation properties.
	int num_par = 10240;

	SimuParams test_params(num_par, //Number of particles.							keep
		num_par,					//Number of particles per stream.				keep
		8,						//Sequence repeat time.								delete
		0.5,						//Sequence echo time.							delete
		0.001,						//Simulation timestep.							keep
		0,							//Number of particles to track continual, individual magnetization.				maybe not//n_mags_track
		Vector3(0, 0, 1),			//Initial magnetization vector.					keep
		Vector3(0, 0, 0.001),		//Main B0 field direction / strength.			keep
		65,							//(vertical) resolution.						keep(need to modify it)
		65,							//(horizontal) resolution.						keep(ntmi)
		5,							//(vertical) FOV.								keep()
		5,							//(horizontal) FOV.								keep()
		1.005,																		//dont need it anymore
		_num										//dont need it anymore
		);

	TRASE_Params test_TRASE(&test_params);

	Coil_Ideal test_coil;
	GRE test_sequence(&test_params);


	Lattice test_lattice(3.0, 3.0, 0.5, 100.0, 100.0, 0, 8);
	Scanner test_scanner(test_sequence, test_coil, test_params, test_lattice,test_TRASE);


	Cylinder_XY test_primitive(Vector3(-1, -1, 0), 0.5, 0.2, 0.5, 1.0, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive);
	Cylinder_XY test_primitive_2(Vector3(1, -1, 0), 0.5, 0.2, 0.5, 1.0, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive_2);
	Cylinder_XY test_primitive_nose(Vector3(0, 0, 0), 0.5, 0.2, 0.5, 1.0, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive_nose);
	Cylinder_XY test_primitive_mouth(Vector3(-1.25, 1, 0), 0.25, 0.2, 0.5, 1.0, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive_mouth);
	Cylinder_XY test_primitive_mouth_2(Vector3(-0.60, 1.25, 0), 0.25, 0.2, 0.5, 1.0, 0.00001, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive_mouth_2);
	Cylinder_XY test_primitive_mouth_3(Vector3(0, 1.4, 0), 0.25, 0.2, 0.5, 1.0, 0.00002, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive_mouth_3);
	Cylinder_XY test_primitive_mouth_4(Vector3(0.60, 1.25, 0), 0.25, 0.2, 0.5, 1.0, 0.00003, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive_mouth_4);
	Cylinder_XY test_primitive_mouth_5(Vector3(1.25, 1, 0), 0.25, 0.2, 0.5, 1.0, 0.00004, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive_mouth_5);

	//Run the scan!
	test_scanner.scan();

	hipDeviceSynchronize();
	hipDeviceReset();

}

int main(){

	for (double i = 25; i < 26;i++){

		iteration(i);

		wait(15);
	}


	return 0;
}
