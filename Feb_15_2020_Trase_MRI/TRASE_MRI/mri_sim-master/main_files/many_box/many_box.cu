#include "hip/hip_runtime.h"
//Main simulator library.
#include "master_def.h"

//Specific coil, sequence... for this simulation.
#include <iostream>
#include "sequence/GRE.cuh"
#include "coil/coil_ideal.cuh"
#include "scanner/scanner.cuh"
#include "primitives/CylinderXY.cuh"
#include "primitives/Box.cuh"
#include "params/simuParams.cuh"
#include "util/recorder.h"
#include "util/vector3.cuh"


#include <time.h>
#include "params/TRASE_Params.cuh"

void wait ( int seconds )
{
  clock_t endwait;
  endwait = clock () + seconds * CLOCKS_PER_SEC ;
  while (clock() < endwait) {}
}


void iteration(real _num){

	//Simulation properties.
	int num_par = 10240;

	SimuParams test_params(num_par, //Number of particles.
		num_par,					//Number of particles per stream.
		8,						//Sequence repeat time.
		0.5,						//Sequence echo time.
		0.001,						//Simulation timestep.
		0,							//Number of particles to track continual, individual magnetization.
		Vector3(0, 0, 1),			//Initial magnetization vector.
		Vector3(0, 0, 0.001),		//Main B0 field direction / strength.
		65,							//(vertical) resolution.
		65,							//(horizontal) resolution.
		5,							//(vertical) FOV.
		5,							//(horizontal) FOV.
		1.005,
		_num
		);

	TRASE_Params test_TRASE(&test_params);


	Coil_Ideal test_coil;
	GRE test_sequence(&test_params);



	Lattice test_lattice(3.0*3, 3.0*3, 0.5*3, 100.0, 100.0, 0, 3*8);
	Scanner test_scanner(test_sequence, test_coil, test_params, test_lattice,test_TRASE);


//new 24_box

	Box test_primitive1(Vector3(-0.65,1.3,0.0), 1.0,1.0,1.0, 2.0, 4.0, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive1);
	Box test_primitive2(Vector3(-0.65,0.0,0.0), 1.0,1.0,1.0, 2.0, 4.0, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive2);
	Box test_primitive3(Vector3(-0.65,-1.3,0.0), 1.0,1.0,1.0, 2.0, 4.0, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive3);

	Box test_primitive4(Vector3(-0.65+(-1.3*1),1.3,0.0), 1.0,1.0,1.0, 2.0, 4.0, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive4);
	Box test_primitive5(Vector3(-0.65+(-1.3*1),0.0,0.0), 1.0,1.0,1.0, 2.0, 4.0, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive5);
	Box test_primitive6(Vector3(-0.65+(-1.3*1),-1.3,0.0), 1.0,1.0,1.0, 2.0, 4.0, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive6);

	Box test_primitive7(Vector3(-0.65+(-1.3*2),1.3,0.0), 1.0,1.0,1.0, 2.0, 4.0, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive7);
	Box test_primitive8(Vector3(-0.65+(-1.3*2),0.0,0.0), 1.0,1.0,1.0, 2.0, 4.0, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive8);
	Box test_primitive9(Vector3(-0.65+(-1.3*2),-1.3,0.0), 1.0,1.0,1.0, 2.0, 4.0, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive9);

	Box test_primitive10(Vector3(-0.65+(-1.3*3),1.3,0.0), 1.0,1.0,1.0, 2.0, 4.0, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive10);
	Box test_primitive11(Vector3(-0.65+(-1.3*3),0.0,0.0), 1.0,1.0,1.0, 2.0, 4.0, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive11);
	Box test_primitive12(Vector3(-0.65+(-1.3*3),-1.3,0.0), 1.0,1.0,1.0, 2.0, 4.0, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive12);













	Box test_primitive13(Vector3(0.65+(1.3*0),1.3,0.0), 1.0,1.0,1.0, 2.0, 4.0, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive13);
	Box test_primitive14(Vector3(0.65+(1.3*0),0.0,0.0), 1.0,1.0,1.0, 2.0, 4.0, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive14);
	Box test_primitive15(Vector3(0.65+(1.3*0),-1.3,0.0), 1.0,1.0,1.0, 2.0, 4.0, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive15);

	Box test_primitive16(Vector3(0.65+(1.3*1),1.3,0.0), 1.0,1.0,1.0, 2.0, 4.0, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive16);
	Box test_primitive17(Vector3(0.65+(1.3*1),0.0,0.0), 1.0,1.0,1.0, 2.0, 4.0, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive17);
	Box test_primitive18(Vector3(0.65+(1.3*1),-1.3,0.0), 1.0,1.0,1.0, 2.0, 4.0, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive18);

	Box test_primitive19(Vector3(0.65+(1.3*2),1.3,0.0), 1.0,1.0,1.0, 2.0, 4.0, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive19);
	Box test_primitive20(Vector3(0.65+(1.3*2),0.0,0.0), 1.0,1.0,1.0, 2.0, 4.0, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive20);
	Box test_primitive21(Vector3(0.65+(1.3*2),-1.3,0.0), 1.0,1.0,1.0, 2.0, 4.0, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive21);

	Box test_primitive22(Vector3(0.65+(1.3*3),1.3,0.0), 1.0,1.0,1.0, 2.0, 4.0, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive22);
	Box test_primitive23(Vector3(0.65+(1.3*3),0.0,0.0), 1.0,1.0,1.0, 2.0, 4.0, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive23);
	Box test_primitive24(Vector3(0.65+(1.3*3),-1.3,0.0), 1.0,1.0,1.0, 2.0, 4.0, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive24);


	//
	test_scanner.scan();
	//


	//Post simulation commands.
	test_scanner.acqs[0]->save_signal("signal");
	test_scanner.acqs[0]->save_tracked("test_track");
	hipDeviceSynchronize();
	hipDeviceReset();

	cout<<"End Here@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@\n"<<endl;


}

int main(){
	//g=25 is better than 30
	for (real i = 18; i < 19;){

		iteration(0);

		i+=100;
		wait(15);
	}


	return 0;
}
