#include "hip/hip_runtime.h"
#include "TRASE.cuh"
#include "pulses.cuh"



__global__ void TRASE_GPU(Sequence** obj_ptr, SimuParams* par, int phase_enc_offset = 0, int _local_res_x = 0);



__device__ __host__ TRASE::TRASE(SimuParams* par, int _phase_enc_offset, int _local_res_x) : Sequence(1,par->TRASE_total, phase_enc_offset, _local_res_x, par){


	phase_enc_offset = _phase_enc_offset;
	readSteps = local_res_x * par->res_y;

	T_res_x = par->res_x;
	T_res_y = par->res_y;


	real pulse_duration = 0.2;					//0.025
	pulse_gap = 0.00;				//0.01
	readFactor = (pulse_gap + pulse_duration)/par->timestep;



//	real FOV_ratio=1;
//	real dg = 0.65/FOV_ratio;



//those variables are for saving the initial echo-train time
	time_gap = 0.1;
	//time_gap is time between each each train
	int array_index = 0;
	real initial_time = 0.0;


	int onedex = 2;
	int num_ratio = 0;
/////A-C-(AC)-(AB)-A









//GIVE THE SIMULATOR STEPS
		steps = (initial_time/par->timestep);
		printf("number of simulator steps %d(JUNYAO)\n",steps);



#ifndef __CUDA_ARCH__
	safe_cuda(hipMalloc(&dev_ptr, sizeof(TRASE**)));
	TRASE_GPU << <1, 1 >> >(dev_ptr, par->devPointer, _phase_enc_offset, _local_res_x);
#ifdef ALLOC_G
	make_tensors();
#endif
#endif
}


__device__ __host__ Vector3 TRASE::getK(int readStep) const{
	int k_y = readStep % par->res_y;
	int k_x = readStep / par->res_y;

	return Vector3(k_x, k_y, 0);
}




__device__ __host__ int TRASE::get_k_start() const{
	return phase_enc_offset*par->res_y;
}

__device__ __host__ int TRASE::get_k_end() const{
	return (local_res_x+phase_enc_offset) * par->res_y;
}




__host__ const Sequence* TRASE::getSubSequences(int i) const{
	if (parallel)
		return &sub_seq[i];
	else
		return this;
}




__device__ __host__ real TRASE::getReadStart(int start) const{

	return pulse[start]->start - pulse_gap/2;
}

__device__ __host__ real TRASE::getReadFinish(int end) const{

	return pulse[end]->end + (time_gap);
}




//what is the point of this function?
__global__ void TRASE_GPU(Sequence** obj_ptr, SimuParams* par, int _phase_enc_offset, int _local_res_x){
	*obj_ptr = new TRASE(par, _phase_enc_offset, _local_res_x);
}

