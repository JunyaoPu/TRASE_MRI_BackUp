#include "hip/hip_runtime.h"
#include "pulses.cuh"

__device__ __host__ Pulse::Pulse(real start, real duration, real strength, Vector3 bDir, Vector3 rDir)
: start(start), duration(duration), strength(strength), bDir(bDir), rDir(rDir){
	end = start + duration;
#ifndef __CUDA_ARCH__
	printf("Pulse start: %f\nPulse end: %f\n================\n", start, end);
#endif
}

/////////////////////////////////////////////////////////TRASE
__device__ __host__ Pulse::Pulse(real start, real duration, Vector3 _g, real angle, Vector3 B0)
: Pulse(start, duration, 0, Vector3(0,0,0), Vector3(0,0,0)){
}


//sub_pulse for TRASE
__device__ __host__ RFflip::RFflip(real start, real duration, Vector3 _g, real angle, Vector3 B0)
:Pulse(start, duration, _g, angle,B0){
//	printf("The function is called \n\n\n");

	strength = angle / fabs(GAMMA*duration);
	w = B0.magnitude() * GAMMA;
	g=_g;
	pulse_switch =1;
}


//excited pulse
__device__ __host__ RFflip_EXC::RFflip_EXC(real start, real duration, Vector3 _g, real angle, Vector3 B0)
:Pulse(start, duration, _g, angle,B0){
	strength = angle / fabs(GAMMA*duration);
	w = B0.magnitude() * GAMMA;
	g=_g;
	pulse_switch =0;
}


//////////////////////
__device__ __host__ RFflip_X::RFflip_X(real start, real duration, Vector3 _g, real angle, Vector3 B0)
:Pulse(start, duration, _g, angle,B0){
	strength = angle / fabs(GAMMA*duration);
	w = B0.magnitude() * GAMMA;
	g=_g;
	pulse_switch =2;
}



__device__ __host__ RFflip_Y::RFflip_Y(real start, real duration, Vector3 _g, real angle, Vector3 B0)
:Pulse(start, duration, _g, angle,B0){
	strength = angle / fabs(GAMMA*duration);
	w = B0.magnitude() * GAMMA;
	g=_g;
	pulse_switch =3;
}



__device__ __host__ real trap(real local_strength, real ramp, real start, real end, real local_time){
	if (local_time - start <= ramp)
		return (1 / ramp)*(local_time-start) * local_strength;
	else if (local_time - start > ramp && local_time <= (end - ramp))
		return local_strength;
	else
		return (1 - (1 / ramp)*(local_time - end + ramp))*local_strength;
}
