#include "hip/hip_runtime.h"
#include "GRE.cuh"
#include "pulses.cuh"



__global__ void GRE_GPU(Sequence** obj_ptr, SimuParams* par, int phase_enc_offset = 0, int _local_res_x = 0);



__device__ __host__ GRE::GRE(SimuParams* par, int _phase_enc_offset, int _local_res_x) : Sequence(1,par->TRASE_total, phase_enc_offset, _local_res_x, par){

	printf("the value of the phase and local res x is %d, %d\n",_phase_enc_offset,_phase_enc_offset);


	phase_enc_offset = _phase_enc_offset;
	readSteps = local_res_x * par->res_y;

	T_res_x = par->res_x;
	T_res_y = par->res_y;


	pulse_duration = 0.2;					//0.025
	//pulse_gap is the time between each pulse
	pulse_gap = 0.0;				//0.01

	//readFactor = (pulse_gap + pulse_duration)/par->timestep;	//read between pulse_gap
	readFactor = (pulse_duration)/par->timestep;				//read immediately after the pulse


	Vector3 A(0,0,0);
	Vector3 B(0,0,0);
	Vector3 C(0,0,0);

	//time_gap is time between each train
	time_gap = 0.1;


	real initial_time = 0.0;
	int onedex = 2;
	int num_ratio = 0;

	int last_pulse_index = 0;





/////A-C-(AC)-(AB)-A			take int(N/2)+2 signals
	for(int num_TR = 0; num_TR<(int)(par->res_x/2);num_TR++)
	{
		onedex = 2;
		pulse[0+num_ratio] = new RFflip_EXC(0.0+initial_time, pulse_duration, A,(PI/2), par-> B0);					//11		//RF_GradientX_A(0, 0.01, Vector3(0,0,1),PI/2, par-> B0);
		//read data
		pulse[1+num_ratio] = new RFflip_Y(pulse[num_ratio]->end + pulse_gap, pulse_duration, C,(par->ratio*PI), par-> B0);								//RF_GradientX_C(pulse[0]->start + (TRASE_TE/2), 0.01, Vector3(0,0,1),PI, par-> B0);

		for(int i =0; i < num_TR; i++)
		{
			pulse[onedex+num_ratio] = new RFflip(pulse[onedex+num_ratio-1]->end + pulse_gap, pulse_duration, A,(par->ratio*PI), par-> B0);			//12		//RF_GradientX_A(pulse[onedex-1]->start + TRASE_TE, 0.01, Vector3(0,0,1),PI, par-> B0);

			pulse[onedex+1+num_ratio] = new RFflip_Y(pulse[onedex+num_ratio]->end + pulse_gap, pulse_duration, C,(par->ratio*PI), par-> B0);		//13		//RF_GradientX_C(pulse[onedex]->start + TRASE_TE, 0.01, Vector3(0,0,1),PI, par-> B0);

			onedex+=2;
		}
		for(int i =0; i < (int)(par->res_y/2); i++)					//here take N/2 signal from the simulator
		{
			pulse[onedex+num_ratio] = new RFflip(pulse[onedex+num_ratio-1]->end + pulse_gap, pulse_duration, A,(par->ratio*PI), par-> B0);						//RF_GradientX_A(pulse[onedex-1]->start + TRASE_TE, 0.01, Vector3(0,0,1),PI, par-> B0);
			pulse[onedex+1+num_ratio] = new RFflip_X(pulse[onedex+num_ratio]->end + pulse_gap, pulse_duration, B,(par->ratio*PI), par-> B0);					//RF_GradientX_A(pulse[onedex]->start + TRASE_TE, 0.01, Vector3(0,0,1),PI, par-> B0);
			onedex+=2;
		}
		pulse[onedex+num_ratio] = new RFflip(pulse[onedex+num_ratio-1]->end + pulse_gap, pulse_duration, A,(par->ratio*PI), par-> B0);
		//record the last read pulse
		TRASE_last[last_pulse_index] = onedex+num_ratio;
		TRASE_first[last_pulse_index] = TRASE_last[last_pulse_index] - (par->res_y);
		last_pulse_index++;


		initial_time = pulse[onedex+num_ratio]->end + time_gap;


		num_ratio +=(2+(num_TR*2)+((int)(par->res_y/2))*2)+1;
	}








//A-C-(AC)-(BA)-A		take int(N/2) signals
	for(int num_TR = 0; num_TR < (par->res_x)-1-(int)(par->res_x/2);num_TR++)
	{
		onedex = 2;
		pulse[0+num_ratio] = new RFflip_EXC(0+initial_time, pulse_duration, A,(PI/2), par-> B0);					//11
		pulse[1+num_ratio] = new RFflip_Y(pulse[num_ratio]->end + pulse_gap, pulse_duration, C,(par->ratio*PI), par-> B0);
		for(int i =0; i < num_TR; i++)
		{
			pulse[onedex+num_ratio] = new RFflip(pulse[onedex+num_ratio-1]->end + pulse_gap, pulse_duration, A,(par->ratio*PI), par-> B0);			//12
			pulse[onedex+1+num_ratio] =  new RFflip_Y(pulse[onedex+num_ratio]->end + pulse_gap, pulse_duration, C,(par->ratio*PI), par-> B0);		//13
			onedex+=2;
		}

		for(int i =0; i < (int)(par->res_y/2); i++)					//here take N/2 signal from the simulator
		{
			pulse[onedex+num_ratio] = new RFflip_X(pulse[onedex+num_ratio-1]->end + pulse_gap, pulse_duration, B,(par->ratio*PI), par-> B0);
			//read data
			pulse[onedex+1+num_ratio] = new RFflip(pulse[onedex+num_ratio]->end + pulse_gap, pulse_duration, A,(par->ratio*PI), par-> B0);
			onedex+=2;
		}

		//record the last read pulse
		TRASE_last[last_pulse_index] = onedex+1+num_ratio - 2;
		TRASE_first[last_pulse_index] = TRASE_last[last_pulse_index] - (par->res_y -2);
		last_pulse_index++;


		initial_time = pulse[(onedex-2)+1+num_ratio]->end + time_gap;

		num_ratio +=(2+(num_TR*2)+((int)(par->res_y/2))*2);
	}











//k-space center	A-(AB)-A   take int(N/2)+1 signals
	onedex = 1;
	pulse[0+num_ratio] = new RFflip_EXC(0+initial_time, pulse_duration, A,(PI/2), par-> B0);


	//read data

	for(int i =0; i < (int)(par->res_y/2); i++)
	{
		//printf("The last pulse is %d\n",onedex+num_ratio);
		pulse[onedex+num_ratio] = new RFflip(pulse[onedex+num_ratio-1]->end + pulse_gap, pulse_duration, A,(par->ratio*PI), par-> B0);
		pulse[onedex+1+num_ratio] = new RFflip_X(pulse[onedex+num_ratio]->end + pulse_gap, pulse_duration,B,(par->ratio*PI), par-> B0);
		onedex +=2;
	}
	pulse[onedex+num_ratio] = new RFflip(pulse[onedex+num_ratio-1]->end + pulse_gap, pulse_duration, A,(par->ratio*PI), par-> B0);



	//record the last read pulse
	TRASE_last[last_pulse_index] = onedex+num_ratio;
	TRASE_first[last_pulse_index] = TRASE_last[last_pulse_index] - (par->res_y -1);

	initial_time = pulse[onedex+num_ratio]->end + time_gap;

//GIVE THE SIMULATOR STEPS
	steps = (initial_time/par->timestep);
	printf("number of simulator steps %d(JUNYAO)\n",steps);


#ifndef __CUDA_ARCH__
	safe_cuda(hipMalloc(&dev_ptr, sizeof(GRE**)));
	GRE_GPU << <1, 1 >> >(dev_ptr, par->devPointer, _phase_enc_offset, _local_res_x);
#ifdef ALLOC_G
	make_tensors();
#endif
#endif
}


__device__ __host__ Vector3 GRE::getK(int readStep) const{
	int k_y = readStep % par->res_y;
	int k_x = readStep / par->res_y;

	return Vector3(k_x, k_y, 0);
}




__device__ __host__ int GRE::get_k_start() const{
	return phase_enc_offset*par->res_y;
}

__device__ __host__ int GRE::get_k_end() const{
	return (local_res_x+phase_enc_offset) * par->res_y;
}




__host__ const Sequence* GRE::getSubSequences(int i) const{
	if (parallel)
		return &sub_seq[i];
	else
		return this;
}

__device__ __host__ real GRE::getReadStart(int start) const{

	return pulse[start]->start - pulse_gap;
}

__device__ __host__ real GRE::getReadFinish(int end) const{

	return pulse[end]->end + (time_gap);
}




//what is the point of this function?
__global__ void GRE_GPU(Sequence** obj_ptr, SimuParams* par, int _phase_enc_offset, int _local_res_x){
	*obj_ptr = new GRE(par, _phase_enc_offset, _local_res_x);
}

