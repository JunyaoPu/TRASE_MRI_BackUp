#include "hip/hip_runtime.h"
//Main simulator library.
#include "master_def.h"

//Specific coil, sequence... for this simulation.
#include <iostream>
#include "sequence/GRE.cuh"
#include "coil/coil_ideal.cuh"
#include "scanner/scanner.cuh"
#include "primitives/CylinderXY.cuh"
#include "primitives/Box.cuh"
#include "params/simuParams.cuh"
#include "util/recorder.h"
#include "util/vector3.cuh"


#include <time.h>
#include "params/TRASE_Params.cuh"

#include "primitives/Box.cuh"

void wait ( int seconds )
{
  clock_t endwait;
  endwait = clock () + seconds * CLOCKS_PER_SEC ;
  while (clock() < endwait) {}
}


void iteration(real _num){

	//Simulation properties.
	int num_par = 10240;

	SimuParams test_params(num_par, //Number of particles.
		num_par,					//Number of particles per stream.
		8,						//Sequence repeat time.
		0.5,						//Sequence echo time.
		0.001,						//Simulation timestep.
		0,							//Number of particles to track continual, individual magnetization.
		Vector3(0, 0, 1),			//Initial magnetization vector.
		Vector3(0, 0, 0.001),		//Main B0 field direction / strength.
		65,							//(vertical) resolution.
		65,							//(horizontal) resolution.
		5,							//(vertical) FOV.
		5,							//(horizontal) FOV.
		1.005
		);

	TRASE_Params test_TRASE(&test_params);
	Coil_Ideal test_coil;
	GRE test_sequence(&test_params);




//two samples
////////////////////////////////////////////////////////////////////////////////////////////////////////////
/*
	Lattice test_lattice(5.0, 5.0, 5.0, 0.0, 0.0, 0, 2);
	Scanner test_scanner(test_sequence, test_coil, test_params, test_lattice,test_TRASE);

	Cylinder_XY test_primitive(Vector3(-1, 0, 0), 0.9, 0.2, 9999.0, 9999.0, 0.0/1000.0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive);
	Cylinder_XY test_primitive_1(Vector3(1, 0, 0), 0.9, 0.2, 9999.0, 9999.0, 0.0 , 1, 0, num_par);		//the diffusion coefficient must be a float point
	test_scanner.add_primitive(test_primitive_1);
*/




/*
//one sample
						//x,y,z
	//Lattice test_lattice(3.0, 3.0, 0.5, 0.0, 0.0, 0, 1);
	Lattice test_lattice(5.0, 5.0, 5.0, 0.0, 0.0, 0, 1);

	Scanner test_scanner(test_sequence, test_coil, test_params, test_lattice,test_TRASE);
//	Cylinder_XY test_primitive(Vector3(0.0,0.0,0.0), 2, 2, 0.0, 0.0, 0.0/1000.0, 1, 0, num_par);

	Cylinder_XY test_primitive(Vector3(0.0,0.0,0.0), 4.0, 2.0, 9999.0*1000, 9999.0*1000, 0.0/1000.0, 1, 0, num_par);			//1.386

	test_scanner.add_primitive(test_primitive);
*/


//////////////////////////////////////////////////////////////////////////////////////////////////////////



	//new 24_box
	real relaxation = 9999.00;

	Lattice test_lattice(3.0*3, 3.0*3, 0.5*3, 100.0, 100.0, 0, 3*8);
	Scanner test_scanner(test_sequence, test_coil, test_params, test_lattice,test_TRASE);

	Box test_primitive1(Vector3(-0.65,1.3,0.0), 1.0,1.0,1.0, relaxation, relaxation, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive1);
	Box test_primitive2(Vector3(-0.65,0.0,0.0), 1.0,1.0,1.0, relaxation, relaxation, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive2);
	Box test_primitive3(Vector3(-0.65,-1.3,0.0), 1.0,1.0,1.0, relaxation, relaxation, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive3);

	Box test_primitive4(Vector3(-0.65+(-1.3*1),1.3,0.0), 1.0,1.0,1.0, relaxation, relaxation, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive4);
	Box test_primitive5(Vector3(-0.65+(-1.3*1),0.0,0.0), 1.0,1.0,1.0, relaxation, relaxation, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive5);
	Box test_primitive6(Vector3(-0.65+(-1.3*1),-1.3,0.0), 1.0,1.0,1.0, relaxation, relaxation, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive6);

	Box test_primitive7(Vector3(-0.65+(-1.3*2),1.3,0.0), 1.0,1.0,1.0, relaxation, relaxation, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive7);
	Box test_primitive8(Vector3(-0.65+(-1.3*2),0.0,0.0), 1.0,1.0,1.0, relaxation, relaxation, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive8);
	Box test_primitive9(Vector3(-0.65+(-1.3*2),-1.3,0.0), 1.0,1.0,1.0, relaxation, relaxation, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive9);

	Box test_primitive10(Vector3(-0.65+(-1.3*3),1.3,0.0), 1.0,1.0,1.0, relaxation, relaxation, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive10);
	Box test_primitive11(Vector3(-0.65+(-1.3*3),0.0,0.0), 1.0,1.0,1.0, relaxation, relaxation, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive11);
	Box test_primitive12(Vector3(-0.65+(-1.3*3),-1.3,0.0), 1.0,1.0,1.0, relaxation, relaxation, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive12);

	Box test_primitive13(Vector3(0.65+(1.3*0),1.3,0.0), 1.0,1.0,1.0, relaxation, relaxation, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive13);
	Box test_primitive14(Vector3(0.65+(1.3*0),0.0,0.0), 1.0,1.0,1.0, relaxation, relaxation, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive14);
	Box test_primitive15(Vector3(0.65+(1.3*0),-1.3,0.0), 1.0,1.0,1.0, relaxation, relaxation, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive15);

	Box test_primitive16(Vector3(0.65+(1.3*1),1.3,0.0), 1.0,1.0,1.0, relaxation, relaxation, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive16);
	Box test_primitive17(Vector3(0.65+(1.3*1),0.0,0.0), 1.0,1.0,1.0, relaxation, relaxation, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive17);
	Box test_primitive18(Vector3(0.65+(1.3*1),-1.3,0.0), 1.0,1.0,1.0, relaxation, relaxation, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive18);

	Box test_primitive19(Vector3(0.65+(1.3*2),1.3,0.0), 1.0,1.0,1.0, relaxation, relaxation, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive19);
	Box test_primitive20(Vector3(0.65+(1.3*2),0.0,0.0), 1.0,1.0,1.0, relaxation, relaxation, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive20);
	Box test_primitive21(Vector3(0.65+(1.3*2),-1.3,0.0), 1.0,1.0,1.0, relaxation, relaxation, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive21);

	Box test_primitive22(Vector3(0.65+(1.3*3),1.3,0.0), 1.0,1.0,1.0, relaxation, relaxation, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive22);
	Box test_primitive23(Vector3(0.65+(1.3*3),0.0,0.0), 1.0,1.0,1.0, relaxation, relaxation, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive23);
	Box test_primitive24(Vector3(0.65+(1.3*3),-1.3,0.0), 1.0,1.0,1.0, relaxation, relaxation, 0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive24);










	//single sample
	/*
/////////////////////////////////////////////////////////////////////////////////////////////////////////
	Scanner test_scanner(test_sequence, test_coil, test_params,test_TRASE);

	Cylinder_XY test_primitive(Vector3(0.0,0.0,0.0), 4.0, 2.0, 9999.0, 9999.0, 0.0/1000.0, 0, 0, num_par);			//1.386
	test_scanner.add_primitive(test_primitive);
/////////////////////////////////////////////////////////////////////////////////////////////////////////
	*/



	//GPU kernel
	test_scanner.scan();

	//CPU kernel
//	test_scanner.scanCPU();

	hipDeviceSynchronize();
	hipDeviceReset();


}

int main(){

	iteration(0);


	return 0;
}
