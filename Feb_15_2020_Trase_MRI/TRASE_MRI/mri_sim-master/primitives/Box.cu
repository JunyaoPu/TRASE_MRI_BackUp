#include "hip/hip_runtime.h"

#include <stdio.h>
/*
	Box. Augments old versions with a cylinder model that has a finite length.
	Old Box changed to infBox to better reflect its model. Based on infBox
	but with length parameter.
	Added: July 18, 2016
	Author: Michael Honke (based on original Box, now infBox, by Trevor Vincent).
	*/
#include "Box.cuh"

__global__ void Box_GPU(Box** obj_ptr, Vector3 _center, real _l_half,real _w_half, real _d_half, real _T2, real _T1, real _D, int _region, real _permeability, int num_particles, real _eps);

__device__ __host__ Box::Box(Vector3 _center, real _l_half,real _w_half, real _d_half, real _T2, real _T1, real _D, int _region, real _permeability, int _num_particles, real _eps){
#ifndef __CUDA_ARCH__
	hipMalloc(&dev_ptr, sizeof(Box**));
	Box_GPU << <1, 1 >> >(dev_ptr, _center, _l_half,_w_half,_d_half, _T2, _T1, _D, _region, _permeability, num_particles, _eps);
#endif
	num_particles = _num_particles;
	center = _center;
	l_half = _l_half;
	w_half = _w_half;
	d_half = _d_half;
	D = _D;
	T2 = _T2;
	T1 = _T1;
	region = _region;
	cylEPS = _eps;

	length = d_half;

	right_end = center.z + length / 2;
	left_end = center.z - length / 2;
}








//__device__ __host__ ~Box(){};

__host__ Primitive** Box::devPointer() const{
	return (Primitive**) dev_ptr;
}

//this will need to be changed once we go to full generality
__device__ Vector3 Box::unifRand(hiprandState localState) const{

	Vector3 r;
	do{
		r.x = l_half*(2.0*hiprand_uniform(&localState) - 1) / 2.0 + center.x;
		r.y = w_half*(2.0*hiprand_uniform(&localState) - 1) / 2.0 + center.y;
		r.z = d_half*(2.0*hiprand_uniform(&localState) - 1) / 2.0 + center.z;
	} while (!inside(r));

	return r;
}

__host__ Vector3 Box::unifRandCPU() const{

	Vector3 r;
	do{
		r.x = l_half*(2.0*unifRandCPP() - 1) / 2.0 + center.x;
		r.y = w_half*(2.0*unifRandCPP() - 1) / 2.0 + center.y;
		r.z = d_half*(2.0*unifRandCPP() - 1) / 2.0 + center.z;
	} while (!inside(r));

	return r;
}


__device__ __host__ bool Box::inside(const Vector3 & r) const{

	return (abs(r.x - center.x) < l_half / 2)&& (abs(r.y - center.y) < w_half / 2) && (abs(r.z - center.z) < d_half / 2);
}

__device__ __host__ bool Box::inside_on(const Vector3 & r) const{

	const real cyc_x = abs(r.x - center.x);
	const real cyc_y = abs(r.y - center.y);
	const real cyc_z = abs(r.z - center.z);
	return ((cyc_x < l_half / 2 - EPSILON)&& (cyc_y < w_half / 2 - EPSILON)&& (cyc_z < d_half / 2 - EPSILON));
}

__device__ __host__ bool Box::inside_on_side(const Vector3 & r) const{

	const real cyc_x = abs(r.x - center.x);
	const real cyc_y = abs(r.y - center.y);
	const real cyc_z = abs(r.z - center.z);
	return ((cyc_x < l_half / 2 - EPSILON)&& (cyc_y < w_half / 2 - EPSILON) && (cyc_z < d_half / 2 - EPSILON));
}

__device__ __host__ bool Box::inside_on_end(const Vector3 & r) const{

	const real cyc_x = abs(r.x - center.x);
	const real cyc_y = abs(r.y - center.y);
	const real cyc_z = abs(r.z - center.z);
	return ((real_equal(cyc_x, l_half/2, cylEPS))&& (real_equal(cyc_y, w_half/2, cylEPS)) && (real_equal(cyc_z, d_half/2, cylEPS)));
}

__device__ __host__ bool Box::inside(real x, real y, real z) const{

	return (abs(x - center.x) < l_half / 2)&& (abs(y - center.y) < w_half / 2)&& (abs(z - center.z) < d_half / 2);
}











/*
	intersect: Determines if a particle has hit the side of the object.
	*/

__device__ __host__ bool Box::intersect(const Vector3 & ri, const Vector3 & rf, real & v, Vector3 & n) const{

	printf("Box::intersect is called########################################(JUNYAO)\n");



	//Particle is inside the cylinder and might hit the left end or particle is to the left of the cylinder and might hit the left end.
	if ((rf.z < left_end && inside_on(ri) == true) || (ri.z < left_end && rf.z > left_end)){
		return intersect_end(ri, rf, v, n, Vector3(0, 0, -1), left_end);
	} //Particle is inside the cylinder and might hit the right end or particle is to the right of the cylinder and might hit the right end.
	else if ((rf.z > right_end && inside_on(ri) == true) || (ri.z > right_end && rf.z < right_end)){
		return intersect_end(ri, rf, v, n, Vector3(0, 0, 1), right_end);
	} //Particle cannot possibly hit either end, but check if it hits the side.
	else {
		return intersect_side(ri, rf, v, n);
	}
}


__device__ __host__ bool Box::intersect_end(const Vector3 & ri, const Vector3 & rf, real & v, Vector3 & n, const Vector3 & n_dir, real end_z) const{
	Vector3 dr = rf - ri;
	//printf("End: %f\n", end_z);
	//printf("rf: %f %f %f\nri:%f %f %f\n", rf.x, rf.y, rf.z, ri.x, ri.y, ri.z);
	real line_param = (end_z - ri.z) / dr.z;
	//printf("Line_param = %f\n", line_param);
	real y_intersect = ri.y + dr.y * line_param;
	real x_intersect = ri.x + dr.x * line_param;
	//printf("y, x: %f %f\n", y_intersect, x_intersect);
	real rho = sqrt(pow(x_intersect - center.x, 2) + pow(y_intersect - center.y, 2));
	//printf("rho: %f\n", rho);

	//Particle is within the area of the cylinder end.
	if (rho <= radius){
		v = abs(end_z - ri.z) / abs(rf.z - ri.z);
		if (real_equal(v, 0.0, cylEPS)){//Then particle started on the wall and should move a full distance.
			return intersect_side(ri, rf, v, n);
		}
		Vector3 temp = (rf - ri)*v + ri;
		//printf("rfc: %f %f %f\n", temp.x, temp.y, temp.z);
		//printf("v: %f\n", v);
		n = n_dir;
		return true;
	} //Particle is outside the area of the cylinder end, but could possibly hit a cylinder side.
	else {
		return intersect_side(ri, rf, v, n);
	}
}

__device__ __host__ bool Box::intersect_side(const Vector3 & ri, const Vector3 & rf, real & v, Vector3 & n) const{

	Vector3 dr = rf - ri;
	real step_mag = dr.magnitude();

	// real a = dr.x*dr.x + dr.y*dr.y;
	// real b = 2*ri.x*dr.x + 2*ri.y*dr.y;
	// real c = ri.x*ri.x + ri.y*ri.y - radius*radius;

	real a = dr.x*dr.x + dr.y*dr.y;
	real b = 2.0*ri.x*dr.x - 2.0*dr.x*center.x + 2.0*ri.y*dr.y - 2.0*dr.y*center.y;
	real c = ri.x*ri.x + ri.y*ri.y - 2.0*ri.x*center.x - 2.0*ri.y*center.y + center.x*center.x + center.y*center.y - radius*radius;

	real q = -.5*(b + sgn(b)*sqrt(b*b - 4 * a*c));
	real root1 = q / a;
	real root2 = c / q;

	bool s1 = (root1 > 0.0 && root1 < 1.0 && b*b>4 * a*c && !real_equal(root1*step_mag, 0.0, cylEPS));
	bool s2 = (root2 > 0.0 && root2 < 1.0 && b*b>4 * a*c && !real_equal(root2*step_mag, 0.0, cylEPS));
	bool s3 = (fabs(root1) < fabs(root2));

	if ((s1 && s2 && s3) || (s1 && !s2)){
		v = root1;
		n = getNormalSide((rf - ri)*v + ri);
		return true;
	}

	else if ((s1 && s2 && !s3) || (s2 && !s1)){
		v = root2;
		n = getNormalSide((rf - ri)*v + ri);
		return true;
	}

	else {
		return false;
	}

}

//here r is a point on the surface
//Needs to be updated for length parameter
__device__ __host__ Vector3 Box::getNormal(const Vector3 & r) const{

	printf("Box::getNormal is called########################################(JUNYAO)\n");


	if (inside_on_side(r))
		return getNormalSide(r);
	else
		return getNormalEnd(r);
}

__device__ __host__ Vector3 Box::getNormalSide(const Vector3 & r) const{
	double n_x = r.x - center.x;
	double n_y = r.y - center.y;
	double mag = sqrt(n_x*n_x + n_y*n_y);
	return Vector3(n_x / mag, n_y / mag, 0.0);
}

__device__ __host__ Vector3 Box::getNormalEnd(const Vector3 & r) const{
	if (r.z < center.z)
		return Vector3(0.0,0.0,-1.0);
	else
		return Vector3(0.0,0.0,1.0);
}









__device__ __host__ real Box::getRadius() const{

	printf("Box::getRadius is called########################################(JUNYAO)\n");

	return radius;

}

__device__ __host__ int Box::getRegion(const Vector3 & r) const{

//	printf("Box::getRegion is called########################################(JUNYAO)\n");

	return region;
}

__device__ __host__  real Box::getT2(const Vector3 & r) const{

	printf("Box::getT2 is called########################################(JUNYAO)\n");

	if (inside(r)){
		return T2;
	}
	return -1.0;
}

__device__ __host__ real Box::getT2() const{
	return T2;
}

__device__ __host__ real Box::getT1() const{
	return T1;
}

__device__ __host__ real Box::getD(const Vector3 & r) const{


	printf("Box::getD is called########################################(JUNYAO)\n");
	if (inside(r)){
		return D;
	}
	return -1.0;
}

__device__ __host__ real Box::getD() const{
	return D;
}

__device__ __host__ real Box::getPermeability() const{
	return permeability;
}

__device__ __host__ Vector3 Box::getCenter() const{
	return Vector3(center.x, center.y, center.z);
}

__host__ void Box::setCenter(Vector3 v){
	center.x = v.x;
	center.y = v.y;
	center.z = v.z;
}

__host__ void Box::setRadius(real _r){

	printf("Box::setRadius is called########################################(JUNYAO)\n");
	radius = _r;
}

__host__ void Box::setEPS(real _cylEPS){
	printf("Box::setEPS is called########################################(JUNYAO)\n");
	cylEPS = _cylEPS;
}

__host__ void Box::setRegion(int _region){
	printf("Box::setRegion is called########################################(JUNYAO)\n");
	region = _region;
}

__device__ __host__ int Box::getRegion(){
	return region;
}

__device__ void Box::randUnif(Vector3 & r, hiprandState & localState) const{
	printf("Box::randUnif is called########################################(JUNYAO)\n");

	do {
		r = Vector3((2.0*hiprand_uniform(&localState) - 1.0)*radius, (2.0*hiprand_uniform(&localState) - 1.0)*radius, (2.0*hiprand_uniform(&localState) - 1.0)*length / 2) + getCenter();
	} while (!inside(r));
}

__host__ void Box::randUnif(Vector3 & r) const{
	printf("Box::randUnif2 is called########################################(JUNYAO)\n");

	do {
		r = Vector3((2.0*unifRandCPP() - 1.0)*radius, (2.0*unifRandCPP() - 1.0)*radius, length*(2.0*unifRandCPP() - 1) / 2.0) + getCenter();
	} while (!inside(r));
}






__global__ void Box_GPU(Box** obj_ptr, Vector3 _center, real _l_half,real _w_half, real _d_half, real _T2, real _T1, real _D, int _region, real _permeability, int num_particles, real _eps){
	if (threadIdx.x == 0 && blockIdx.x == 0)
		*obj_ptr = new Box(_center, _l_half,_w_half,_d_half, _T2, _T1, _D, _region, _permeability, num_particles, _eps);
}
