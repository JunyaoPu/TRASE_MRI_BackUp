#include "hip/hip_runtime.h"
#include "Update_Mag.cuh"



__device__ void Update_Mag(real &Mx,
		real &My,
		real &Mz,
		real w,
		real time,
		real time_step,
		real strength90,
		real strength180,
		real dg,
		Vector3 B0,
		Vector3 r,
		int pulse_pick,
		real T1,
		real T2
		){
		Vector3 TRASE_M,k1,k2,k3,k4,finaldM;
		Vector3 v1,v2,v3;
		Vector3 G;
		real pulse_strength;


		TRASE_M.x = Mx;
		TRASE_M.y = My;
		TRASE_M.z = Mz;

		real trig_num1=w*time;
		real trig_num2=w*(time + .5*time_step);
		real trig_num3=w*(time + time_step);
















//here is doing	(*C)->getField(*B, r, tn)+B0

		if(pulse_pick==0){			//Exciting pulse

			pulse_strength =2*strength90*cos(trig_num1)*0.995;
			v1=Vector3(pulse_strength, 0, 0)+B0;

			pulse_strength =2*strength90*cos(trig_num2)*0.995;
			v2=Vector3(pulse_strength, 0, 0)+B0;

			pulse_strength =2*strength90*cos(trig_num3)*0.995;
			v3=Vector3(pulse_strength, 0, 0)+B0;


		}else if(pulse_pick == 1){		//refocusing pulse with no phase


			pulse_strength =2*strength180*cos(trig_num1)*0.995;
			G=Vector3(0, pulse_strength, 0);
			v1=G+B0;

			pulse_strength =2*strength180*cos(trig_num2)*0.995;
			G=Vector3(0, pulse_strength, 0);
			v2=G+B0;

			pulse_strength =2*strength180*cos(trig_num3)*0.995;
			G=Vector3(0, pulse_strength, 0);
			v3=G+B0;


			/*
		if(pulse_pick==0){			//Exciting pulse

			pulse_strength =2*strength90*cos(trig_num1)*0.995;
			v1=Vector3(strength90*cos(w*time), -strength90*sin(w*time), 0)+B0;
			pulse_strength =2*strength90*cos(trig_num2)*0.995;
			v2=Vector3(strength90*cos(w*time), -strength90*sin(w*time), 0)+B0;
			pulse_strength =2*strength90*cos(trig_num3)*0.995;
			v3=Vector3(strength90*cos(w*time), -strength90*sin(w*time), 0)+B0;


		}else if(pulse_pick == 1){		//refocusing pulse with no phase


			pulse_strength =2*strength180*cos(trig_num1)*0.995;
			G=Vector3(-strength180*cos(w*time+PI/2), strength180*sin(w*time+PI/2), 0);
			v1=G+B0;

			pulse_strength =2*strength180*cos(trig_num2)*0.995;
			G=Vector3(-strength180*cos(w*time+PI/2), strength180*sin(w*time+PI/2), 0);
			v2=G+B0;

			pulse_strength =2*strength180*cos(trig_num3)*0.995;
			G=Vector3(-strength180*cos(w*time+PI/2), strength180*sin(w*time+PI/2), 0);
			v3=G+B0;
			*/



		}else if(pulse_pick ==2){		//refocusing pulse with X phase

			real pulse_X= (dg)*(r.x);

			pulse_strength =2*strength180*cos(trig_num1)*0.995;
			G=Vector3(pulse_strength*sin(pulse_X), pulse_strength*cos(pulse_X), 0);
			v1=G+B0;

			pulse_strength =2*strength180*cos(trig_num2)*0.995;
			G=Vector3(pulse_strength*sin(pulse_X), pulse_strength*cos(pulse_X), 0);
			v2=G+B0;

			pulse_strength =2*strength180*cos(trig_num3)*0.995;
			G=Vector3(pulse_strength*sin(pulse_X), pulse_strength*cos(pulse_X), 0);
			v3=G+B0;

		}else{	//refocusing pulse with Y phase				//here reduced 4s for the simulator
			real pulse_Y=(dg)*(r.y);

			pulse_strength =2*strength180*cos(trig_num1)*0.995;
			G=Vector3(pulse_strength*sin(pulse_Y), pulse_strength*cos(pulse_Y), 0);
			v1=G+B0;

			pulse_strength =2*strength180*cos(trig_num2)*0.995;
			G=Vector3(pulse_strength*sin(pulse_Y), pulse_strength*cos(pulse_Y), 0);
			v2=G+B0;

			pulse_strength =2*strength180*cos(trig_num3)*0.995;
			G=Vector3(pulse_strength*sin(pulse_Y), pulse_strength*cos(pulse_Y), 0);
			v3=G+B0;
		}








//no relaxation
		/*
		k1=(TRASE_M % v1 )*GAMMA*time_step;
		k2 = ((TRASE_M + k1*.5)% v2 )*GAMMA*time_step;
		k3 = ((TRASE_M + k2*.5)% v2)*GAMMA*time_step;
		k4 = ((TRASE_M + k3)%v3 )*GAMMA*time_step;
		*/




//with relaxation
		k1=(((TRASE_M % v1 )*GAMMA)- Vector3(Mx / T2, My / T2, (Mz - 1.0) / T1))*time_step;
		k2 = ((((TRASE_M + k1*.5)% v2 )*GAMMA)- Vector3(Mx / T2, My / T2, (Mz - 1.0) / T1))*time_step;
		k3 = ((((TRASE_M + k2*.5)% v2)*GAMMA)- Vector3(Mx / T2, My / T2, (Mz - 1.0) / T1))*time_step;
		k4 = ((((TRASE_M + k3)%v3 )*GAMMA)- Vector3(Mx / T2, My / T2, (Mz - 1.0) / T1))*time_step;


		finaldM = (k1 + k2*2.0 + k3*2.0 + k4)*(1.0 / 6.0);
		Mx += finaldM.x;
		My += finaldM.y;
		Mz += finaldM.z;

}











__device__ Vector3 pulse_Exc(real strength, real trig_num){
	//tip on + x-axis(apply on  -y-axis)
	return Vector3(-strength*sin(trig_num), -strength*cos(trig_num), 0);

}

__device__ Vector3 pulse_ref(real strength, real trig_num){

	//apply on  +x-axis
	return Vector3(strength*cos(trig_num), -strength*sin(trig_num), 0);
}

__device__ Vector3 pulse_axis(real strength, real trig_num, real axis){
	real cos_angle = cos(trig_num);
	real sin_angle = sin(trig_num);

	real cos_gz = cos(axis);
	real sin_gz = sin(axis);

	//on +x axis
	real x_field = cos_angle*strength*cos_gz + sin_angle*strength*sin_gz;
	real y_field = -sin_angle*strength*cos_gz + cos_angle*strength*sin_gz;

	return Vector3(x_field, y_field, 0);



}



