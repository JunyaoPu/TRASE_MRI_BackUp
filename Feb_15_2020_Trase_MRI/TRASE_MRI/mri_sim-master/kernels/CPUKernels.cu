#include "hip/hip_runtime.h"
#include "CPUkernels.cuh"
#include <iostream>

void updateWalkersMagCPU(SimuParams *par,
		Primitive* basis,
		const Sequence* B,
		Coil* coil,
		int n_mags_track,
		real *signal_x,
		real *signal_y,
		real *signal_z,

		real *d_seg_time,
		int *d_read_start,
		int *d_read_end,
		int *d_pulse_switch
						) {

//overall variable
	printf("the CPU kernel is called\n\n\n\n\n");
	real w = (par->B0).z * GAMMA;
	real dg=0.22;						//14*14 FOV
	real strength90=(B)->TRASE_90_CPU();
	real strength180=(B)->TRASE_180_CPU();
	Vector3 B0=par->B0;								//constant
	real time_step= par->timestep;					//constant




	int d_seg=0;
	int pulse_num=0;
	int pulse_pick= 0;		// 0 , 1 , 2, 3


	//loop the number of particles
	for (int tid = 0; tid < par->number_of_particles; tid++) {
//	for (int tid = 0; tid < 1; tid++) {
		printf("Current particle on CPU is: %d\n", tid);


		real Mx = 0;
		real My = 0;
		real Mz = 0;



		//reset the particle variable
		d_seg=0;
		pulse_num=0;
		pulse_pick= 0;		// 0 , 1 , 2, 3


		//pre-diffusion
		real phi, theta;
		real speed = sqrt(6.0 * basis->getD() / par->timestep);



		//generate the position
		Vector3 r = basis->unifRandCPU();
		//printf("%f, %f, %f\n", r.x, r.y, r.z);


		//initialize the M
		Mx = 0;
		My = 0;
		Mz = 1;







		//loop is here
		////////////////////////TRASE
		int s = 0;
		Vector3 TRASE_M,k1,k2,k3,k4,finaldM;
		Vector3 v1,v2,v3;
		///////////////////////TRASE
		for (int i = 0; i < (B)->TRASE_getSteps_CPU(); i++) {


			real time = i * par->timestep;





/*//no diffusion
		//Diffusion
			Vector3 ri = r;
			phi = 2.0 * PI * unifRandCPP();
			theta = acos(2.0 * unifRandCPP() - 1);
			r += Vector3(speed * par->timestep * sin(theta) * cos(phi),
					speed * par->timestep * sin(theta) * sin(phi),
					speed * par->timestep * cos(theta));
#if defined SPECULAR_REFLECTION
			boundaryNormalCPU(ri,r, speed, basis, par->timestep);
#else
			if (!basis->inside(r)) {
				r = ri;
			}
#endif
*/




///////////////////////////////////////////////////////////////////////////////////////////TRASE
		//Reset the mag at the first pulse of each echo_train
				if(time < d_seg_time[d_seg]){
				}else{
					d_seg++;

					Mx = 0;
					My = 0;
					Mz = 1;
				}

		//update the pulse number
				if(pulse_num < par->TRASE_total-1){
					pulse_num = (B)->update_pulse_num_CPU(time,pulse_num);
				}


				pulse_pick = d_pulse_switch[pulse_num];
/////////////////////////////////////////////////////////////////////////////////////////TRASE


/*
#if defined RK4_RELAXATION


		TRASE_M.x=Mx;
		TRASE_M.y=My;
		TRASE_M.z=Mz;

		k1=((TRASE_M%((B)->TRASE_getG_CPU( r, time,pulse_num) + B0))*GAMMA)*time_step;								//1143
		k2 = (((TRASE_M + k1*.5)%((B)->TRASE_getG_CPU( r, time + .5*time_step,pulse_num) + B0))*GAMMA)*time_step;	//1784
		k3 = (((TRASE_M + k2*.5)% ((B)->TRASE_getG_CPU( r, time + .5*time_step, pulse_num) + B0))*GAMMA)*time_step;//14
		k4 = (((TRASE_M + k3)%((B)->TRASE_getG_CPU( r, time + time_step, pulse_num) + B0))*GAMMA)*time_step;		//9

		finaldM = (k1 + k2*2.0 + k3*2.0 + k4)*(1.0 / 6.0);

		Mx += finaldM.x;
		My += finaldM.y;
		Mz += finaldM.z;

#endif
*/






//////////////////////////////////////////////////////////////////////////////////////////////TRASE
			//X-Y Plane		x-y field only
			if(((B)->TRASE_No_Relaxation_CPU(time,pulse_num))){


				TRASE_M.x=Mx;
				TRASE_M.y=My;
				TRASE_M.z=Mz;

				Vector3 G;
				real pulse_strength;


				real trig_num1=w*time;
				real trig_num2=w*(time + .5*time_step);
				real trig_num3=w*(time + time_step);



				if(pulse_pick==0){			//Exciting pulse

					pulse_strength =2*strength90*cos(trig_num1)*0.995;
					v1=Vector3(pulse_strength, 0, 0)+B0;

					pulse_strength =2*strength90*cos(trig_num2)*0.995;
					v2=Vector3(pulse_strength, 0, 0)+B0;

					pulse_strength =2*strength90*cos(trig_num3)*0.995;
					v3=Vector3(pulse_strength, 0, 0)+B0;


				}else if(pulse_pick == 1){		//refocusing pulse with no phase


					pulse_strength =2*strength180*cos(trig_num1)*0.995;
					G=Vector3(0, pulse_strength, 0);
					v1=G+B0;

					pulse_strength =2*strength180*cos(trig_num2)*0.995;
					G=Vector3(0, pulse_strength, 0);
					v2=G+B0;

					pulse_strength =2*strength180*cos(trig_num3)*0.995;
					G=Vector3(0, pulse_strength, 0);
					v3=G+B0;





				}else if(pulse_pick ==2){		//refocusing pulse with X phase

					real pulse_X= (dg)*(r.x);

					pulse_strength =2*strength180*cos(trig_num1)*0.995;
					G=Vector3(pulse_strength*sin(pulse_X), pulse_strength*cos(pulse_X), 0);
					v1=G+B0;

					pulse_strength =2*strength180*cos(trig_num2)*0.995;
					G=Vector3(pulse_strength*sin(pulse_X), pulse_strength*cos(pulse_X), 0);
					v2=G+B0;

					pulse_strength =2*strength180*cos(trig_num3)*0.995;
					G=Vector3(pulse_strength*sin(pulse_X), pulse_strength*cos(pulse_X), 0);
					v3=G+B0;

				}else{	//refocusing pulse with Y phase				//here reduced 4s for the simulator
					real pulse_Y=(dg)*(r.y);

					pulse_strength =2*strength180*cos(trig_num1)*0.995;
					G=Vector3(pulse_strength*sin(pulse_Y), pulse_strength*cos(pulse_Y), 0);
					v1=G+B0;

					pulse_strength =2*strength180*cos(trig_num2)*0.995;
					G=Vector3(pulse_strength*sin(pulse_Y), pulse_strength*cos(pulse_Y), 0);
					v2=G+B0;

					pulse_strength =2*strength180*cos(trig_num3)*0.995;
					G=Vector3(pulse_strength*sin(pulse_Y), pulse_strength*cos(pulse_Y), 0);
					v3=G+B0;
				}

				k1=( TRASE_M % v1 )*GAMMA*time_step;
	 			k2 = ( (TRASE_M + k1*.5)% v2 )*GAMMA*time_step;
				k3 = ((TRASE_M + k2*.5)% v2)*GAMMA*time_step;
				k4 = ( (TRASE_M + k3)%v3 )*GAMMA*time_step;


	 			finaldM = (k1 + k2*2.0 + k3*2.0 + k4)*(1.0 / 6.0);
				Mx += finaldM.x;
				My += finaldM.y;
				Mz += finaldM.z;

				}
///////////////////////////////////////////////////////////////////////////////////////TRASE







//////////////////////////////////////////////////////////////////////////////////////TRASE
			if (time >= (B)->TRASE_getReadStart_CPU(d_read_start[d_seg]) &&
				time <= (B)->TRASE_getReadFinish_CPU(d_read_end[d_seg]) &&
				((int)(i - (B)->TRASE_getReadStart_CPU(d_read_start[d_seg])/time_step))!=0&&
				( (int)(i - (B)->TRASE_getReadStart_CPU(d_read_start[d_seg])/time_step)) % (B)->TRASE_getReadFactor_CPU() == 0){
/////////////////////////////////////////////////////////////////////////////////////TRASE

				//Save the total signal from this block to global memory for later summation.
				signal_x[s] += Mx * cos(w * time) - My * sin(w * time);
				signal_y[s] += Mx * sin(w * time) + My * cos(w * time);
				signal_z[s] += Mz;
				s++;
			}

		}

	}

}
