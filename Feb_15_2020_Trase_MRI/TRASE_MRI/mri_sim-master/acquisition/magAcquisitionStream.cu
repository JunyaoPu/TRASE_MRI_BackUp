#include "magAcquisitionStream.cuh"
#include "../kernels/kernelMagLattice.cuh"


///////////////////////////////////////////////////////optimization
//single sample
__host__ Scan::Scan(
	magAcquisition* acq,
	SimuParams* par,
	TRASE_Params* TRASE_par,
	const Sequence* host_seq,
	const Primitive* basis,
	const Coil** coil,
	int devNum,
	const std::vector<int> devSMP,
	hipStream_t stream
	):
	acq(acq),
	par(par),
	TRASE_par(TRASE_par),
	host_seq(host_seq),
	basis(basis),
	coil(coil),
	devNum(devNum),
	devSMP(&devSMP),
	stream(stream),
	measurements(par->measurements),
	number_of_particles(par->particles_per_stream)
{
	num_blocks = par->blocks;
	//signal of one block
	signal_x.malloc(host_seq->getReadSteps()*num_blocks, stream);
	signal_y.malloc(host_seq->getReadSteps()*num_blocks, stream);
	signal_z.malloc(host_seq->getReadSteps()*num_blocks, stream);

	//signal of steps
	signal_x_total.malloc(host_seq->getReadSteps(), stream);
	signal_y_total.malloc(host_seq->getReadSteps(), stream);
	signal_z_total.malloc(host_seq->getReadSteps(), stream);


	dev_states.malloc(number_of_particles, stream);
	dev_par.malloc(stream);
	dev_par = *par;

	safe_cuda(hipGetLastError(), "Malloc");
	hipStreamSynchronize(stream);
	dev_par.copyToDevice();
	safe_cuda(hipGetLastError(), "Malloc2");
	setup_kernel <<< num_blocks, SIM_THREADS, 0, stream>>> (dev_states.getPointer(), par->getSeed());
	safe_cuda(hipGetLastError(), "Setup");
}






//multiple sample
__host__ Scan::Scan(
	magAcquisition* acq,
	SimuParams* par,

	TRASE_Params* TRASE_par,


	const Sequence* host_seq,
	Lattice* lattice,
	Primitive*** basis_dev_pointers,
	const Coil** coil,
	int devNum,
	const std::vector<int> devSMP,
	hipStream_t stream
	):
	acq(acq),
	par(par),

	TRASE_par(TRASE_par),

	host_seq(host_seq),
	lattice(lattice),
	basis_dev_pointers(basis_dev_pointers),
	coil(coil),
	devNum(devNum),
	devSMP(&devSMP),
	stream(stream),
	measurements(par->measurements),
	number_of_particles(par->particles_per_stream)
{
	size_t size_basis_dev_pointers = lattice->getBasisSize() * sizeof(Primitive**);
	hipMalloc((void****)&basis_dev_pointers_pointer, size_basis_dev_pointers);
	hipMemcpy(basis_dev_pointers_pointer, basis_dev_pointers, size_basis_dev_pointers, hipMemcpyHostToDevice);
	num_blocks = par->blocks;
	signal_x.malloc(host_seq->getReadSteps()*num_blocks, stream);
	signal_y.malloc(host_seq->getReadSteps()*num_blocks, stream);
	signal_z.malloc(host_seq->getReadSteps()*num_blocks, stream);
	signal_x_total.malloc(host_seq->getReadSteps(), stream);
	signal_y_total.malloc(host_seq->getReadSteps(), stream);
	signal_z_total.malloc(host_seq->getReadSteps(), stream);
	dev_states.malloc(number_of_particles, stream);
	dev_par.malloc(stream);
	dev_par = *par;

	safe_cuda(hipGetLastError(), "Malloc");
	hipStreamSynchronize(stream);
	dev_par.copyToDevice();
	safe_cuda(hipGetLastError(), "Malloc2");
	setup_kernel <<< num_blocks, SIM_THREADS, 0, stream>>> (dev_states.getPointer(), par->getSeed());
	safe_cuda(hipGetLastError(), "Setup");
}
///////////////////////////////////////////////////////////////////









__host__ void Scan::runScan(){


	//copy data
	/*
	real h_seg_time[par->res_y];
	int h_read_start[par->res_y];
	int h_read_end[par->res_y];
	*/

	int h_last[par->res_y];
	int h_first[par->res_y];

	for (int i=0; i <par->res_y;i++){
		/*
		h_seg_time[i] = host_seq->array[i];
		h_read_start[i] = TRASE_par->read_start[i];
		h_read_end[i] = TRASE_par->read_end[i];
		*/

		h_last[i] = host_seq->TRASE_last[i];
		h_first[i] = host_seq->TRASE_first[i];
	}

	//allocate cuda memory
/*
	real *d_seg_time;
	int *d_read_start;
	int *d_read_end;
	int *d_pulse_switch;
*/
	int *d_last;
	int *d_first;


//	int size = sizeof(real);
	int size_int = sizeof(int);
/*
	hipMalloc((void**)&d_seg_time,(par->res_y)*size);
	hipMalloc((void**)&d_read_start,(par->res_y)*size_int);
	hipMalloc((void**)&d_read_end,(par->res_y)*size_int);
	hipMalloc((void**)&d_pulse_switch,(par->TRASE_total)*size_int);
*/
	hipMalloc((void**)&d_last,(par->res_y)*size_int);
	hipMalloc((void**)&d_first,(par->res_y)*size_int);

/*
	hipMemcpy(d_seg_time,h_seg_time,size*(par->res_y),hipMemcpyHostToDevice);
	hipMemcpy(d_read_start,h_read_start,size_int*(par->res_y),hipMemcpyHostToDevice);
	hipMemcpy(d_read_end,h_read_end,size_int*(par->res_y),hipMemcpyHostToDevice);
	hipMemcpy(d_pulse_switch, TRASE_par->pulse_switch,size_int*(par->TRASE_total),hipMemcpyHostToDevice);
*/
	hipMemcpy(d_last,h_last,size_int*(par->res_y),hipMemcpyHostToDevice);
	hipMemcpy(d_first,h_first,size_int*(par->res_y),hipMemcpyHostToDevice);
/*
	delete []host_seq->array;
	delete []TRASE_par->read_start;
	delete []TRASE_par->read_end;
	delete []TRASE_par->pulse_switch;
*/

//	updateWalkersMag<false, false> << < 1, 1, 0, stream >> > (
	updateWalkersMag<false, false> << < num_blocks, SIM_THREADS, 0, stream >> > (
		dev_par.getPointer(),
		basis->devPointer(),
		host_seq->devPointer(),
		coil,
		dev_states.getPointer(),
		0,//par->n_mags_track,
		0,
		0,
		0,
		signal_x.getPointer(),
		signal_y.getPointer(),
		signal_z.getPointer(),
		d_last,
		d_first
		);


}






__host__ void Scan::run_scan_lattice(){

	int h_last[par->res_y];
	int h_first[par->res_y];

	for (int i=0; i <par->res_y;i++){

		h_last[i] = host_seq->TRASE_last[i];
		h_first[i] = host_seq->TRASE_first[i];
	}

	//allocate cuda memory

	int *d_last;
	int *d_first;


//	int size = sizeof(real);
	int size_int = sizeof(int);

	hipMalloc((void**)&d_last,(par->res_y)*size_int);
	hipMalloc((void**)&d_first,(par->res_y)*size_int);

	hipMemcpy(d_last,h_last,size_int*(par->res_y),hipMemcpyHostToDevice);
	hipMemcpy(d_first,h_first,size_int*(par->res_y),hipMemcpyHostToDevice);

//	update_walkers_lattice_mag<false, false> << < 1, 1, 0, stream >> > (
	update_walkers_lattice_mag<false, false> << < num_blocks, SIM_THREADS, 0, stream >> > (
		dev_par.getPointer(),
		lattice->devPointer(),
		basis_dev_pointers_pointer,
		host_seq->devPointer(),
		coil,
		dev_states.getPointer(),
		0,//par->n_mags_track,
		0,
		0,
		0,
		signal_x.getPointer(),
		signal_y.getPointer(),
		signal_z.getPointer(),
		d_last,
		d_first
		);


}

__host__ void Scan::saveScan(){
	int threads_sum = 512;
	signal_x.sum(signal_x_total, threads_sum, NUM_SM, host_seq->getReadSteps(), num_blocks, stream);
	signal_y.sum(signal_y_total, threads_sum, NUM_SM, host_seq->getReadSteps(), num_blocks, stream);
	signal_z.sum(signal_z_total, threads_sum, NUM_SM, host_seq->getReadSteps(), num_blocks, stream);
	safe_cuda(hipDeviceSynchronize());
	signal_x_total.copyFromDevice();
	signal_y_total.copyFromDevice();
	signal_z_total.copyFromDevice();
	safe_cuda(hipDeviceSynchronize());
	signal_x_total.copyTo(acq->get_signal_x());
	signal_y_total.copyTo(acq->get_signal_y());
	signal_z_total.copyTo(acq->get_signal_z());
}
