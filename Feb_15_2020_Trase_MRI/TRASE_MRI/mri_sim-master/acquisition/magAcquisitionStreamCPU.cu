#include "magAcquisitionStreamCPU.cuh"

__host__ ScanCPU::ScanCPU(){}
__host__ ScanCPU::ScanCPU(
	magAcquisition* acq,
	SimuParams* par,

	TRASE_Params* TRASE_par,

	const Sequence* host_seq,
	Primitive* basis,
	Coil* coil





	):
	acq(acq),
	par(par),

	TRASE_par(TRASE_par),

	host_seq(host_seq),
	basis(basis),
	coil(coil),
	measurements(par->measurements),
	number_of_particles(par->particles_per_stream)
{
	signal_x = new real[host_seq->getReadSteps()];
	signal_y = new real[host_seq->getReadSteps()];
	signal_z = new real[host_seq->getReadSteps()];



}

__host__ void ScanCPU::runScan(){



	real h_seg_time[par->res_y];

	int h_read_start[par->res_y];
	int h_read_end[par->res_y];

//	int h_pulse_switch[10];

	for (int i=0; i <par->res_y;i++){

		h_seg_time[i] = host_seq->array[i];

		h_read_start[i] = TRASE_par->read_start[i];
		h_read_end[i] = TRASE_par->read_end[i];
	}



	updateWalkersMagCPU(
		par,
		basis,
		host_seq,
		coil,
		par->n_mags_track,
		signal_x,
		signal_y,
		signal_z,



		h_seg_time,
		h_read_start,
		h_read_end,
		TRASE_par->pulse_switch

		);
}

__host__ void ScanCPU::saveScan(){
	for (int i = 0; i < host_seq->getReadSteps(); i++){
		acq->get_signal_x()[i] = (signal_x[i]);
		acq->get_signal_y()[i] = (signal_y[i]);
		acq->get_signal_z()[i] = (signal_z[i]);
	}
}
