#include <chrono>
#include "scanner.cuh"
#include "../primitives/lattice.cuh"



Scanner::Scanner(
	Sequence& sequence,
	Coil& coil,
	SimuParams& params,

	TRASE_Params& TRASE_params)

	:
	sequence(&sequence),
	coil(&coil),
	params(&params),
	TRASE_params(&TRASE_params)
	{
	scan_k = new kSpace(params.res_x, params.res_y);
	lattice_present = false;
}

Scanner::Scanner(												//multiple samples after optimization
	Sequence& sequence,
	Coil& coil,
	SimuParams& params,
	Lattice& lattice,

	TRASE_Params& TRASE_params)
	:
	sequence(&sequence),
	coil(&coil),
	params(&params),
	lattice(&lattice),
	TRASE_params(&TRASE_params)
	{
	scan_k = new kSpace(params.res_x, params.res_y);
	lattice_present = true;
}




//In the iteration file
bool Scanner::scan(){
	bool scan_success;

	if (lattice_present){
		scan_success = scan_lattice();
	} else {
		scan_success = scan_single_basis();
	}

	return scan_success;
}






bool Scanner::scan_lattice(){
	printf("Building basis pointer array.\n");
	lattice->setBasisSize(primitives.size());
	printf("Detected %d basis objects\n", lattice->getBasisSize());
	Primitive** basis_dev_pointers[lattice->getBasisSize()];


	for (int i = 0; i < primitives.size(); i++){
		basis_dev_pointers[i] = primitives[i]->devPointer();
	}


	hipStream_t streams[sequence->getNSubSequences()];
	printf("Starting scan.\n");
	printf("Number of sub sequences = %d.\n", sequence->getNSubSequences());
	//Scan* scans = new Scan[sequence->getNSubSequences()];
	Scan* scans[sequence->getNSubSequences()];



	for (int i = 0; i< sequence->getNSubSequences(); i++){
		acqs.push_back(new magAcquisition(params, sequence->getSubSequences(i)));
		hipStreamCreate(&streams[i]);
		scans[i] = new Scan(acqs[i], params,TRASE_params, sequence->getSubSequences(i), lattice, basis_dev_pointers, coil->devPointer(), 0, std::vector<int>(15), streams[i]);
	}
	hipDeviceSynchronize();



	long start = std::chrono::duration_cast< std::chrono::milliseconds >(std::chrono::system_clock::now().time_since_epoch()).count();
	for (int i = 0; i < sequence->getNSubSequences(); i++){
		printf("Running scan %d/%d\n",i,sequence->getNSubSequences()-1);




		scans[i]->run_scan_lattice();
	}



	safe_cuda(hipDeviceSynchronize(), "Post Scan\n");
	long end = std::chrono::duration_cast< std::chrono::milliseconds >(std::chrono::system_clock::now().time_since_epoch()).count();
	printf("Simulation Kernel Time: %ld\n", end-start);




	for (int i = 0; i < sequence->getNSubSequences(); i++){
		scans[i]->saveScan();
		hipDeviceSynchronize();
		make_k_space(acqs[i], sequence->getSubSequences(i));
	}

	hipDeviceSynchronize();

	scan_k->get_fft();

	return true;
}



bool Scanner::scan_single_basis(){

	hipStream_t streams[sequence->getNSubSequences()];

	printf("Starting scan.\n");
	printf("Number of sub sequences = %d.\n", sequence->getNSubSequences());

	//initialize the Scan class, each Scan is one CUDA stream
	Scan* scans[sequence->getNSubSequences()];



	//loop the CUDA stream
	for (int i = 0; i< sequence->getNSubSequences(); i++){
		//acqs is mag acquisition function
		acqs.push_back(new magAcquisition(params, sequence->getSubSequences(i)));

		hipStreamCreate(&streams[i]);
		//initialize the scans class for each CUDA stream
		scans[i] = new Scan(acqs[i], params,TRASE_params, sequence->getSubSequences(i), primitives[0], coil->devPointer(), 0, std::vector<int>(15), streams[i]);
	}


	hipDeviceSynchronize();

	//running the scan
	for (int i = 0; i < sequence->getNSubSequences(); i++){

		long start = std::chrono::duration_cast< std::chrono::milliseconds >(std::chrono::system_clock::now().time_since_epoch()).count();

		scans[i]->runScan();

		long end = std::chrono::duration_cast< std::chrono::milliseconds >(std::chrono::system_clock::now().time_since_epoch()).count();
		printf("Simulation Kernel Time: %ld\n", end-start);

	}



	safe_cuda(hipDeviceSynchronize(), "Post Scan\n");


	//here is the post-simulation
	for (int i = 0; i < sequence->getNSubSequences(); i++){

		scans[i]->saveScan();
		hipDeviceSynchronize();
		make_k_space(acqs[i], sequence->getSubSequences(i));

	}
	hipDeviceSynchronize();

	scan_k->get_fft();

	return true;
}






bool Scanner::scanCPU(){
	printf("Starting scan.\n");
	printf("Number of sub sequences = %d.\n", sequence->getNSubSequences());
	//Scan* scans = new Scan[sequence->getNSubSequences()];
	ScanCPU* scans[sequence->getNSubSequences()];

	for (int i = 0; i< sequence->getNSubSequences(); i++){
		acqs.push_back(new magAcquisition(params, sequence->getSubSequences(i)));


		scans[i] = new ScanCPU(acqs[i], params,TRASE_params, sequence->getSubSequences(i), primitives[0], coil);

	}

	long start = std::chrono::duration_cast< std::chrono::milliseconds >(std::chrono::system_clock::now().time_since_epoch()).count();
	for (int i = 0; i < sequence->getNSubSequences(); i++){
		printf("Running scan %d/%d\n",i,sequence->getNSubSequences()-1);



		//run the cpu kernel
		scans[i]->runScan();
	}

	long end = std::chrono::duration_cast< std::chrono::milliseconds >(std::chrono::system_clock::now().time_since_epoch()).count();
	printf("Simulation Kernel Time: %ld\n", end-start);



	for (int i = 0; i < sequence->getNSubSequences(); i++){


		scans[i]->saveScan();
		make_k_space(acqs[i], sequence->getSubSequences(i));
	}

	scan_k->get_fft();

	return true;
}



bool Scanner::add_primitive(Primitive& new_primitive){
	primitives.push_back(&new_primitive);
	
	return true;
}



bool Scanner::make_k_space(magAcquisition *acq, const Sequence *seq){
	int kx;
	int ky;

	for (int i = seq->get_k_start(); i < seq->get_k_end(); i++){
		kx = seq->getK(i).x;
		ky = seq->getK(i).y;
		scan_k->set_Mx(kx, ky, acq->get_signal_x()[i%seq->getReadSteps()]);
		scan_k->set_My(kx, ky, acq->get_signal_y()[i%seq->getReadSteps()]);

	}
//rearrange TRASE k-space and shift the phase
	scan_k->TRASE_rearrangeMx(scan_k->dim_x,scan_k->dim_y);
	scan_k->TRASE_rearrangeMy(scan_k->dim_x,scan_k->dim_y);

	scan_k->TRASE_phase_shiftingMx(scan_k->dim_x,scan_k->dim_y);
	scan_k->TRASE_phase_shiftingMy(scan_k->dim_x,scan_k->dim_y);

	scan_k->TRASE_transferMx();
	scan_k->TRASE_transferMy();

	return true;
}
