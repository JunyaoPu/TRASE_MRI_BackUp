#include"k_space.cuh"

kSpace::kSpace(const int dim_x, const int dim_y) :dim_x(dim_x), dim_y(dim_y){
	host_space = (hipfftDoubleComplex**)malloc(sizeof(hipfftDoubleComplex*)*dim_x);
	host_result = (hipfftDoubleComplex**)malloc(sizeof(hipfftDoubleComplex*)*dim_x);

	host_space[0] = (hipfftDoubleComplex *)malloc(dim_y*dim_x*sizeof(hipfftDoubleComplex));
	host_result[0] = (hipfftDoubleComplex *)malloc(dim_y*dim_x*sizeof(hipfftDoubleComplex));

	for (int i = 1; i < dim_x; i++){
		host_space[i] = host_space[i - 1] + dim_y;
		host_result[i] = host_result[i - 1] + dim_y;
	}

	host_space_pitch = dim_y * sizeof(hipfftDoubleComplex);
	hipMallocPitch(&dev_space, &dev_space_pitch, dim_y*sizeof(hipfftDoubleComplex), dim_x);

///////////////////////////////////////////////////////////////////////////////////////////////////////////////
	Tdim_x=64;
	Tdim_y=64;

	Thost_space = (hipfftDoubleComplex**)malloc(sizeof(hipfftDoubleComplex*)*Tdim_x);
	Thost_result = (hipfftDoubleComplex**)malloc(sizeof(hipfftDoubleComplex*)*Tdim_x);

	Thost_space[0] = (hipfftDoubleComplex *)malloc(Tdim_y*Tdim_x*sizeof(hipfftDoubleComplex));
	Thost_result[0] = (hipfftDoubleComplex *)malloc(Tdim_y*Tdim_x*sizeof(hipfftDoubleComplex));

	for (int i = 1; i < dim_x; i++){
		Thost_space[i] = Thost_space[i - 1] + Tdim_y;
		Thost_result[i] = Thost_result[i - 1] + Tdim_y;
	}

	Thost_space_pitch = Tdim_y * sizeof(hipfftDoubleComplex);
	hipMallocPitch(&Tdev_space, &Tdev_space_pitch, Tdim_y*sizeof(hipfftDoubleComplex), Tdim_x);

}

size_t kSpace::index(int x, int y){
	return y + x*dim_y;
}

double kSpace::get_Mx(int x, int y){
	return host_space[x][y].x;
}

double kSpace::get_My(int x, int y){
	return host_space[x][y].y;
}

void kSpace::set_Mx(int x, int y, double val){
	host_space[x][y].x += val;
}
void kSpace::set_My(int x, int y, double val){
	host_space[x][y].y += val;
}


//////////////////////////////////////////////////////
void kSpace::get_fft(){
	hipMemcpy2D(Tdev_space,
		Tdev_space_pitch,
		Thost_space[0],
		Thost_space_pitch,
		Tdim_y*sizeof(hipfftDoubleComplex),
		Tdim_x,
		hipMemcpyHostToDevice);

	printf("Starting FFT Process\n");
	hipfftHandle plan;
	hipfftPlan2d(&plan, Tdim_x, Tdim_y, HIPFFT_Z2Z);
	printf("Plan Built\n");
	hipfftExecZ2Z(plan, Tdev_space, Tdev_space, HIPFFT_BACKWARD);
	hipDeviceSynchronize();
	hipMemcpy2D(Thost_result[0], Thost_space_pitch, Tdev_space, Tdev_space_pitch,
		Tdim_y * sizeof(hipfftDoubleComplex), Tdim_x, hipMemcpyDeviceToHost);


//save_csv_complex

/*
	recorder image_k_r("k_space_r");
	image_k_r.save_csv_complex<hipfftDoubleComplex>(Thost_space, 'r', Tdim_x, Tdim_y);

	recorder image_k_i("k_space_i");
	image_k_i.save_csv_complex<hipfftDoubleComplex>(Thost_space, 'i', Tdim_x, Tdim_y);




//save_image_complex
	recorder image_k("k_space");
	image_k.save_image_complex<hipfftDoubleComplex>(Thost_space, 'm', Tdim_x, Tdim_y);
	*/
	recorder image("image");
	image.save_image_complex<hipfftDoubleComplex>(Thost_result, 'm', Tdim_x, Tdim_y);

}


/*
void kSpace::get_fft(){
	hipMemcpy2D(dev_space,
		dev_space_pitch,
		host_space[0],
		host_space_pitch,
		dim_y*sizeof(hipfftDoubleComplex),
		dim_x,
		hipMemcpyHostToDevice);

	printf("Starting FFT Process\n");
	hipfftHandle plan;
	hipfftPlan2d(&plan, dim_x, dim_y, HIPFFT_Z2Z);
	printf("Plan Built\n");
	hipfftExecZ2Z(plan, dev_space, dev_space, HIPFFT_BACKWARD);
	hipDeviceSynchronize();
	hipMemcpy2D(host_result[0], host_space_pitch, dev_space, dev_space_pitch,
		dim_y * sizeof(hipfftDoubleComplex), dim_x, hipMemcpyDeviceToHost);

	recorder image_k_r("k_space_r");
	image_k_r.save_csv_complex<hipfftDoubleComplex>(host_space, 'r', dim_x, dim_y);

	recorder image_k_i("k_space_i");
	image_k_i.save_csv_complex<hipfftDoubleComplex>(host_space, 'i', dim_x, dim_y);

	recorder image_k("k_space");
	image_k.save_image_complex<hipfftDoubleComplex>(host_space, 'm', dim_x, dim_y);

	recorder image("image");
	image.save_image_complex<hipfftDoubleComplex>(host_result, 'm', dim_x, dim_y);
}

*/

